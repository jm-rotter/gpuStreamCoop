#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include <thread>
#include <hip/hip_runtime.h>

#define Buffersize 1024

__device__ uint8_t* d_arg_buffer;
__device__ int flag = 1;

__device__ void test() {
    printf("test: d_arg_buffer[0] = %d\n", d_arg_buffer[0]);
}

__global__ void kernel() {
    printf("in kernel\n");
    printf("flag: %d\n", flag);

    while (atomicAdd(&flag, 0)) {
    }

    printf("flag down\n");
    test();    
}

void synchronizer(hipStream_t stream, hipEvent_t event) {
    printf("launched synchronizer\n");

    hipStreamWaitEvent(stream, event, 0);

    int falseval = 0;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(flag), &falseval, sizeof(int), 0, hipMemcpyHostToDevice, stream);
	printf("cleared flag to 0 (async)\n");

    hipStreamSynchronize(stream);

    printf("flag cleared\n");
}

int main() {
    int data = 42;

    hipStream_t stream;
    hipEvent_t event;
    hipStreamCreate(&stream);
    hipEventCreate(&event);

	hipStream_t stream2;
	hipStreamCreate(&stream2);

    uint8_t* dab;
    hipMalloc(&dab, Buffersize);

    int one = 1;
    hipMemcpyToSymbol(HIP_SYMBOL(flag), &one, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_arg_buffer), &dab, sizeof(dab));

    kernel<<<1, 1, 0, stream2>>>();

    hipMemcpyAsync(dab, &data, 1, hipMemcpyHostToDevice, stream);

    hipEventRecord(event, stream);

    std::thread t1(synchronizer, std::ref(stream), std::ref(event));
    t1.join();

    hipStreamSynchronize(stream);
    hipDeviceSynchronize();

    hipFree(dab);
    hipEventDestroy(event);
    hipStreamDestroy(stream);
	cudaStreamDestory(stream2);

    return 0;
}

